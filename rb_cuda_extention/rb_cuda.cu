#include "hip/hip_runtime.h"
// #include<hip/hip_runtime.h>
#include"rb_cuda.h"

__global__ void _calculate(float* d_a, float* d_b, float* d_c, int n, int iters){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx == 7){
        printf("Hello from cuda_0: thread_7 .\n");
    }
    if(idx < n){
        for(int i=0;i<iters;i++)
            d_c[idx] += d_a[idx] + d_b[idx];
    }
    return;
}

void calculate(int n_power, int iters_power){
    int n = 1 << n_power;
    int iters = 1 << iters_power;
    printf("rb_cuda校验 -> n: %d, iters: %d\n", n, iters);
    int block_size = 256;
    int grid_size = ceil(n/block_size);
    int size = n*sizeof(float);
    float* h_a = (float*)malloc(size);
    float* h_b = (float*)malloc(size);
    float* h_c = (float*)malloc(size);

    for(int i=0;i<n;i++){
        h_a[i] = i;
        h_b[i] = i;
    }

    float* d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    std::cout << "Correspond cpu_thread is blocked...\n" << std::endl;

    _calculate<<<grid_size, block_size>>>(d_a, d_b, d_c, n, iters);

    // 检查内核执行错误
    // CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
    hipMemcpy(h_c, d_c, n, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for(int i=0;i<1<<4;i++){
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }
    
    free(h_a);
    free(h_b);
    free(h_c);

    std::cout << "Correspond cpu_thread is running...\n" << std::endl;
    return;
}