#include<hip/hip_runtime.h>
#include<iostream>

__global__ void hello(){
    printf("Hello from cuda_0.\n");
    return;
}

int main(){
    std::cout << "Hello from cpu." << std::endl;
    hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}